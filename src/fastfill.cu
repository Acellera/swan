
#include <hip/hip_runtime.h>
__global__ void swan_fast_fill( uint4 *ptr, int len ) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if( idx<len) {
		ptr[idx] = make_uint4( 0,0,0,0 );
	}
}

__global__ void swan_fast_fill_word( uint *ptr, int len ) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if( idx<len) {
		ptr[idx] = 0;
	}
}




__global__ void canary( int N ) {
//	int idx = threadIdx.x + blockDim.x * blockIdx.x;
//	if( idx < N ) {
//		out[idx] = in[idx];
//	}
}


