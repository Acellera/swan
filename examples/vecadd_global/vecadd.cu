
#include <hip/hip_runtime.h>
__device__ float in1[ 1000 ];
__device__ __constant__ float in2[ 1000 ];

__global__ void vecadd( float *out, int N ) {
	int idx= blockDim.x * blockIdx.x + threadIdx.x;
	if( idx < N ) {
		out[idx] =in1[idx] + in2[idx];
	}
}
